#include "hip/hip_runtime.h"
#include "solver.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <float.h>
#include <stdexcept>

__device__ float compute_energy(const float* J, int N, unsigned int k) {
    float energy = 0.0f;
    for (int i = 0; i < N; ++i) {
        int Si = ((k >> i) & 1) ? 1 : -1;
        for (int j = i + 1; j < N; ++j) {
            int Sj = ((k >> j) & 1) ? 1 : -1;
            energy -= J[i * N + j] * Si * Sj;
        }
    }
    return energy;
}

__global__ void compute_energies_and_min(
    const float* __restrict__ J,
    int N,
    float* __restrict__ energies)
{
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= (1U << (N - 1))) return;

    float energy = compute_energy(J, N, k);
    energies[k] = energy;
}

void launch_energy_kernel(const float* h_J, int N, float*& h_energies) {
    size_t total_k = 1ULL << (N - 1);

    float* d_J;
    float* d_energies;

    hipMalloc(&d_J, sizeof(float) * N * N);
    hipMalloc(&d_energies, sizeof(float) * total_k);

    hipMemcpy(d_J, h_J, sizeof(float) * N * N, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (total_k + threadsPerBlock - 1) / threadsPerBlock;

    compute_energies_and_min<<<blocks, threadsPerBlock>>>(d_J, N, d_energies);
    hipDeviceSynchronize();

    h_energies = new float[total_k];

    hipMemcpy(h_energies, d_energies, sizeof(float) * total_k, hipMemcpyDeviceToHost);

    hipFree(d_J);
    hipFree(d_energies);
}
